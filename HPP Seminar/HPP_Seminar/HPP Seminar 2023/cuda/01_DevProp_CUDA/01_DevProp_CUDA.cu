#include<stdio.h>
#include<hip/hip_runtime.h>

int main(void)
{
	//Function Declarations
	void PrintCUDADeviceProperties(void);

	//code
	PrintCUDADeviceProperties();
}

void PrintCUDADeviceProperties(void)
{
	//code
	printf("Srushti Moghe CUDA Information : \n");
	printf("------------------------\n");

	printf("CUDA Information : \n");
	printf("******============******\n");

	hipError_t ret_cuda_rt;
	int dev_count;
	ret_cuda_rt = hipGetDeviceCount(&dev_count);
	if (ret_cuda_rt != hipSuccess)
	{
		printf("CUDA Runtime API Error - hipGetDeviceCount() Failed due to %s.\n", hipGetErrorString(ret_cuda_rt));
	}
	else if (dev_count == 0)
	{
		printf("There is no CUDA Supported Device On This System.\n");
		return;
	}
	else
	{

		printf("Total Number Of CUDA Supporting GPU Device/Devices On This System : %d\n", dev_count);

		for (int i = 0; i < dev_count; i++)
		{
			hipDeviceProp_t dev_prop;
			int driverVersion = 0, runtimeVersion = 0;
			ret_cuda_rt = hipGetDeviceProperties(&dev_prop, i);
			if (ret_cuda_rt != hipSuccess)
			{
				printf("%s in %s at line %d\n", hipGetErrorString(ret_cuda_rt), __FILE__, __LINE__);
				return;
			}
			printf("\n");
			hipDriverGetVersion(&driverVersion);
			hipRuntimeGetVersion(&runtimeVersion);
			printf("********** CUDA DRIVER and RUNTIME INFORMATION **********\n\n");
			printf("======================================================================================================\n\n");
			printf("SUM CUDA Driver Version          :%d.%d\n", driverVersion / 1000, (driverVersion % 100) / 10);
			printf("SUM CUDA Runtime Version         :%d.%d\n", runtimeVersion / 1000, (runtimeVersion / 100) / 10);
			printf("\n");
			printf("======================================================================================================\n\n");
			printf("********** GPU DEVICE GENERAL INFORMATION **********\n\n");
			printf("======================================================================================================\n\n");
			printf("SUM GPU Device Number                : %d\n", i);
			printf("SUM GPU Device Name                  : %s\n", dev_prop.name);
			printf("SUM GPU Device Compute Capability    :%d.%d\n", dev_prop.major, dev_prop.minor);
			printf("SUM GPU Device Clock rate            :%d\n", dev_prop.clockRate);

			printf("SUM GPU Device Type                  :\n");
			if (dev_prop.integrated)
				printf("Integrated   (On - Board)        :\n");
			else
				printf("Discrete (Card)                  :\n");

			printf("\n");

			printf("********** GPU DEVICE MEMORY INFORMATION **********\n\n");
			printf("======================================================================================================\n\n");
			printf("SUM GPU Device Total Memory						 :  %.0f GB  = %.0f MB == %llu Bytes\n", ((float)dev_prop.totalGlobalMem / 1048576.0f) / 1024.0f, (float)dev_prop.totalGlobalMem / 1048576.0f, (unsigned long long)dev_prop.totalGlobalMem);

			printf("SUM GPU Device Constant Memory                   : %lu Bytes \n", (unsigned long)dev_prop.totalConstMem);
			printf("SUM GPU Device Shared Memory per SMProcessor     : %lu Bytes \n", (unsigned long)dev_prop.sharedMemPerBlock);
			printf("\n");

			printf("**************GPU DEVICE MULTIPROCESSOR INFORMATION ******************\n");
			printf("======================================================================================================\n\n");

			printf("SUM GPU Device Number of SMProcessors                  : %lu Bytes \n", dev_prop.multiProcessorCount);
			printf("SUM GPU Device Number Of Registers Per SMProcessor     : %lu Bytes \n", (unsigned long)dev_prop.regsPerBlock);

			printf("\n");

			printf("************************  GPU DEVICE THREAD INFORMATION  ***********************\n");
			printf("======================================================================================================\n\n");

			printf("SUM GPU Device Maximum Number Of Threads Per SMProcessor : %d\n", dev_prop.maxThreadsPerMultiProcessor);
			printf("SUM GPU Device Maximum Number Of Threads Per Block       :%d\n", dev_prop.maxThreadsPerBlock);

			printf("SUM GPU Device Threads In Warp                           : %d\n", dev_prop.warpSize);

			printf("SUM GPU Device Maximum Thread Dimensions                 : (%d, %d, %d) \n", dev_prop.maxThreadsDim[0], dev_prop.maxThreadsDim[1], dev_prop.maxThreadsDim[2]);

			printf("SUM GPU Device Maximum Grid Dimensions                   : (%d, %d, %d) \n", dev_prop.maxGridSize[0], dev_prop.maxGridSize[1], dev_prop.maxGridSize[2]);

			printf("\n");

			printf("======================================================================================================\n\n");
			printf("*************************GPU DEVICE DRIVER INFORMATION ********************\n");
			printf("======================================================================================================\n\n");
			printf("SUM GPU Device has ECC support				  : %s\n", dev_prop.ECCEnabled ? "Enabled" : "Disabled");

#if defined (WIN32) || defined(_WIN32) || defined(WIN64) || defined( WINE64)
			printf("SUM GPU Device CUDA Driver mode (TCC Or WDDM) :%s \n", dev_prop.tccDriver ? "TCC (Tesla Compute Cluster Driver)" : "WDDM (Windows Display Driver Model)\n");

#endif
			printf("************************************************** END ******************************************************\n\n");
		}
	}
}