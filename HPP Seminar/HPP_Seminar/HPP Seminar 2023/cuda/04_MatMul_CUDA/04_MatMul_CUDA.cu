#include "hip/hip_runtime.h"


#include <stdio.h>
#include <hip/hip_runtime.h>

#include "helper_timer.h"

//Macros

#define BLOCK_WIDTH 32

//const int iNumberOfArrayElements = 5;
//const int iNumberOfArrayElements = 11444777;
//const int iNumberOfArrayElements = 100;

int *hostA = NULL;
int *hostB = NULL;
int *hostC = NULL;
int *gold = NULL;

int *deviceA = NULL;
int *deviceB = NULL;
int *deviceC = NULL;

float timeOnCPU = 0.0f;
float timeOnGPU = 0.0f;

// CUDA kernel
__global__ void matMulGPU(int* A, int* B, int* C, int numARows,int numAColumns,int numBColumns,int numCColumns)
{
	//variable declarations
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int column = blockIdx.x * blockDim.x + threadIdx.x;

	// code
	if ((row < numARows) && column < numAColumns)
	{
		int value = 0.0;
		for (int k = 0; k < numAColumns; k++)
		{
			int a = A[row * numAColumns + k];
			int b = B[k * numBColumns + column];
			value += a * b;
		}
		C[row * numCColumns + column] = value;
	}
}

// entry-point function
int main(void)
{
	// function declarations
	void InitA(int* data, int, int);
	void InitB(int* data, int, int);
	void matMulCPU(int*, int*, int*, int, int, int, int);
	void cleanup(void);

	// variable declarations

	int numARows = BLOCK_WIDTH;
	int numAColumns = BLOCK_WIDTH;
	int numBRows = BLOCK_WIDTH;
	int numBColumns = BLOCK_WIDTH;

	int numCRows = numARows;
	int numCColumns = numBColumns;
	int numGoldRows = numARows;
	int numGoldColumns = numBColumns;

	int sizeA = numARows * numAColumns * sizeof(int);
	int sizeB = numBRows * numBColumns * sizeof(int);
	int sizeC = numCRows * numCColumns * sizeof(int); 

	int sizeGold = numGoldRows * numGoldColumns * sizeof(int);

	hipError_t result = hipSuccess;

	// code

    // host memory allocation
	hostA = (int*)malloc(sizeA);
	if (hostA == NULL)
	{
		printf("SUM: Host Memory allocation is Failed For hostA array");
		cleanup();
		exit(EXIT_FAILURE);
	}

	hostB = (int*)malloc(sizeB);
	if (hostB == NULL)
	{
		printf("SUM: Host Memory allocation is Failed For hostB array");
		cleanup();
		exit(EXIT_FAILURE);
	}
	
	hostC = (int*)malloc(sizeC);
	if (hostC == NULL)
	{
		printf("SUM: Host Memory allocation is Failed For hostC array");
		cleanup();
		exit(EXIT_FAILURE);
	}

	gold = (int*)malloc(sizeGold);
	if (gold == NULL)
	{
		printf("SUM: Host Memory allocation is Failed For gold array");
		cleanup();
		exit(EXIT_FAILURE);
	}
	
	//printing matrix dimensions and sizes
	printf("SUM Output: The Dimensions Of Matrix 'hostA' Are : %d x %d\n", numARows, numAColumns);

	printf("SUM Output: The Dimensions Of Matrix 'hostB' Are : %d x %d\n", numBRows, numBColumns);

	printf("SUM Output: The Dimensions Of Matrix 'hostC' Are : %d x %d\n", numCRows, numCColumns);

	printf("SUM Output: The Dimensions Of Matrix 'gold' Are : %d x %d\n", numGoldRows, numGoldColumns);

	printf("SUM Output: Size of Matrix hostA = %d\n", sizeA);

	printf("SUM Output: Size of Matrix hostB = %d\n", sizeB);

	printf("SUM Output: Size of Matrix hostC = %d\n", sizeC);

	printf("SUM Output: Size of Matrix hostGold = %d\n", sizeGold);

	//Fill source matrices

	InitA(hostA, numARows, numAColumns);
	InitB(hostB, numBRows, numBColumns);

	 // device memory allocation
	  result = hipMalloc((void**)&deviceA, sizeA);
	  if (result != hipSuccess)
	  {
		  printf("SUM: Device Memory allocation is failed	for deviceA array\n");
		  cleanup();
		  exit(EXIT_FAILURE);
	  }

	  result = hipMalloc((void**)&deviceB, sizeB);
	  if (result != hipSuccess)
	  {
		  printf("SUM: Device Memory allocation is failed	for deviceB.\n");
		  cleanup();
		  exit(EXIT_FAILURE);
	  }

	  result = hipMalloc((void**)&deviceC, sizeC);
	  if (result != hipSuccess)
	  {
		  printf("SUM: Device Memory allocation is failed	for deviceC array.\n");
		  cleanup();
		  exit(EXIT_FAILURE);
	  }
	  
	  //Copy data from host arrays into device arrays
	  result = hipMemcpy(deviceA, hostA, sizeA, hipMemcpyHostToDevice);
	  if (result != hipSuccess)
	  {
		  printf("SUM: Host to Device Data copy is Failed for DeviceA array.\n");
		  cleanup();
		  exit(EXIT_FAILURE);
	  }

	  result = hipMemcpy(deviceB, hostB, sizeB, hipMemcpyHostToDevice);
	  if (result != hipSuccess)
	  {
		  printf("SUM: Host to Device Data copy is Failed for DeviceInput2 array.\n");
		  cleanup();
		  exit(EXIT_FAILURE);
	  }

	  //CUDA Kernel Configuration
	  dim3 dimGrid = dim3(ceil((int)numBColumns / (int)BLOCK_WIDTH), ceil((int)numARows / (int)BLOCK_WIDTH), 1);

	  dim3 dimBlock = dim3(BLOCK_WIDTH,BLOCK_WIDTH,1);

	  //CUDA Kernel for Vector Addition
	  StopWatchInterface* timer = NULL;
	  sdkCreateTimer(&timer);
	  sdkStartTimer(&timer);


	  // CUDA kernel for Vector Addition
	  matMulGPU <<<dimGrid, dimBlock >>> (deviceA, deviceB,
		  deviceC, numARows,numAColumns,numBColumns,numCColumns);

	  sdkStopTimer(&timer);
	  timeOnGPU = sdkGetTimerValue(&timer);
	  sdkDeleteTimer(&timer);
	  timer = NULL;

	  // copy data from device matrix into host matrix
	  result = hipMemcpy(hostC, deviceC, sizeC,
		  hipMemcpyDeviceToHost);

	  if (result != hipSuccess)
	  {
		  printf("SUM: Device to Host Data Copy is failed for hostC Matrix.\n");
		  cleanup();
		  exit(EXIT_FAILURE);
	  }

	  //Matrix Multiplication on host
	  matMulCPU(hostA, hostB, gold, numARows,numAColumns,numBColumns,numCColumns);

	  //Comparison

	  //const float epsilon = 0.000001f;
	  int breakValue = -1;
	  bool bAccuracy = true;

	  for (int i = 0; i < numCRows; i++)
	  {
		  float val1 = gold[i];
		  float val2 = hostC[i];

		  if (val1 != val2)
		  {
			  bAccuracy = false;
			  breakValue = i;
			  break;
		  }
	  }

	  char str[128];
	  if (bAccuracy == false)
		  sprintf(str, "SUM Output: Comparison of CPU and GPU Matrix Multiplication is not within accuracy of 0.000001 at array  index %d", breakValue);
	  else
		  sprintf(str, "SUM Output: Comparison of CPU and GPU Matrix Multiplication is within accuracy  of 0.000001");

	  printf("SUM Output: Time Taken for Matrix Multiplication on CPU = %.6f\n\n", timeOnCPU);
	  
	  printf("SUM Output: Time Taken for Matrix Multiplication on GPU = %.6f\n\n", timeOnGPU);
	  printf("%s\n", str);


	  //Cleanup
	  cleanup();

	  return(0);
}

void InitA(int* data, int row,int col)
{
	int num = 1;
	//code
	for (int i = 0; i < row; i++)
	{
		for (int j = 0; j < col; j++)
		{
			*(data + i * col + j) = num;
			num++;
		}
	}
}

void InitB(int* data, int row, int col)
{
	int num = BLOCK_WIDTH;
	//code
	for (int i = 0; i < row; i++)
	{
		for (int j = 0; j < col; j++)
		{
			*(data + i * col + j) = num;
			num--;
		}
	}
}

void matMulCPU( int *A, int* B, int* C,int numARows,int numAColumns,int numBColumns,int numCColumns)
{
	//code
	StopWatchInterface* timer = NULL;
	sdkCreateTimer(&timer);
	sdkStartTimer(&timer);

	for (int i = 0; i < numARows; ++i)
	{
		for (int j = 0; j < numBColumns; ++j)
		{
			int value = 0.0f;
			for (int k = 0; k < numAColumns; ++k)
			{
				int a = A[i * numAColumns + k];
				int b = B[k * numBColumns + j];
				value += a * b;
			}
			C[i * numCColumns + j] = value;
		}
	}

	sdkStopTimer(&timer);
	timeOnCPU = sdkGetTimerValue(&timer);
	sdkDeleteTimer(&timer);
	timer = NULL;
}

void cleanup(void)
{
	//code

	if (gold)
	{
		free(gold);
		gold = NULL;
	}

	if (deviceC)
	{
		hipFree(deviceC);
		deviceC = NULL;
	}

	if (deviceB)
	{
		hipFree(deviceB);
		deviceB = NULL;
	}

	if (deviceA)
	{
		hipFree(deviceA);
		deviceA = NULL;
	}

	if (hostC)
	{
		free(hostC);
		hostC = NULL;
	}

	if (hostB)
	{
		free(hostB);
		hostB = NULL;
	}

	if (hostA)
	{
		free(hostA);
		hostA = NULL;
	}
}