#include "hip/hip_runtime.h"

#include <stdio.h>
#include <hip/hip_runtime.h>

#include "helper_timer.h"


// const int iNumberOfArrayElements = 5;

// const int iNumberOfArrayElements = 11444777;
const int iNumberOfArrayElements = 100;


float* hostInput1 = NULL;
float* hostInput2 = NULL;
float* hostOutput = NULL;
float* gold = NULL;

float* deviceInput1 = NULL;
float* deviceInput2 = NULL;
float* deviceOutput = NULL;

float timeOnCPU = 0.0f;
float timeOnGPU = 0.0f;

// CUDA kernel
__global__ void vecAddGPU(float* in1, float* in2, float* out, int len)

{
	// code
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < len)
	{
		out[i] = in1[i] + in2[i];
	}
}

// entry-point function
int main(void)
{
	// function declarations
	void fillFloatArrayWithRandomNumbers(float*, int);
	void vecAddCPU(const float*, const float*, float*, int);
	void cleanup(void);

	// variable declarations
	int size = iNumberOfArrayElements * sizeof(float);
	hipError_t result = hipSuccess;

	// code

    // host memory allocation
	hostInput1 = (float*)malloc(size);
	if (hostInput1 == NULL)
	{
		printf("SUM: Host Memory allocation is Failed For hostInput1 array");
		cleanup();
		exit(EXIT_FAILURE);
	}

	hostInput2 = (float*)malloc(size);

	if (hostInput2 == NULL)
	{
		printf("SUM: Host Memory allocation is Failed For hostInput2 array");
		cleanup();
		exit(EXIT_FAILURE);
	}
	
	hostOutput = (float*)malloc(size);

	if (hostOutput == NULL)
	{
		printf("SUM: Host Memory allocation is Failed For hostOutput array");
		cleanup();
		exit(EXIT_FAILURE);
	}

	gold = (float*)malloc(size);
	if (gold == NULL)
	{
		printf("SUM: Host Memory allocation is Failed For gold array");
		cleanup();
		exit(EXIT_FAILURE);
	}
	
	// Filling value into host arrays

	fillFloatArrayWithRandomNumbers(hostInput1, iNumberOfArrayElements);
	fillFloatArrayWithRandomNumbers(hostInput2, iNumberOfArrayElements);


	 // device memory allocation
	  result = hipMalloc((void**)&deviceInput1, size);
	  if (result != hipSuccess)
	  {
		  printf("SUM: Device Memory allocation isfailed	for deviceInput1 array\n");
		  cleanup();
		  exit(EXIT_FAILURE);
	  }

	  result = hipMalloc((void**)&deviceInput2, size);
	  if (result != hipSuccess)
	  {
		  printf("SUM: Device Memory allocation isfailed	for deviceInput12array.\n");
		  cleanup();
		  exit(EXIT_FAILURE);
	  }

	  result = hipMalloc((void**)&deviceOutput, size);
	  if (result != hipSuccess)
	  {
		  printf("SUM: Device Memory allocation isfailed	for deviceOutput array.\n");
		  cleanup();
		  exit(EXIT_FAILURE);
	  }
	  
	  // Copy data from host arrays into device arrays
	  result = hipMemcpy(deviceInput1, hostInput1, size, hipMemcpyHostToDevice);
	  if (result != hipSuccess)
	  {
		  printf("SUM: Host to Device Data copy is Failed for DeviceInput1 array.\n");
		  cleanup();
		  exit(EXIT_FAILURE);
	  }

	  result = hipMemcpy(deviceInput2, hostInput2, size, hipMemcpyHostToDevice);
	  if (result != hipSuccess)
	  {
		  printf("SUM: Host to Device Data copy is Failed for DeviceInput2 array.\n");
		  cleanup();
		  exit(EXIT_FAILURE);
	  }

	  // CUDA Kernel Configuration
	  dim3 dimGrid = dim3((int)ceil((float)iNumberOfArrayElements / 256.0f), 1, 1);


	  dim3 dimBlock = dim3(256, 1, 1);

	  // CUDA Kernel for Vector Addition
	  StopWatchInterface* timer = NULL;
	  sdkCreateTimer(&timer);
	  sdkStartTimer(&timer);


	  // CUDA kernel for Vector Addition
	  vecAddGPU <<<dimGrid, dimBlock >>> (deviceInput1, deviceInput2,
		  deviceOutput, iNumberOfArrayElements);

	  sdkStopTimer(&timer);
	  timeOnGPU = sdkGetTimerValue(&timer);
	  sdkDeleteTimer(&timer);
	  timer = NULL;

	  // copy data from device array into host array
	  result = hipMemcpy(hostOutput, deviceOutput, size,
		  hipMemcpyDeviceToHost);

	  if (result != hipSuccess)
	  {
		  printf("SUM: Device to Host Data Copy is failed for hostOutput array.\n");

		  cleanup();
		  exit(EXIT_FAILURE);
	  }

	  // Vector addition on host
	  vecAddCPU(hostInput1, hostInput2, gold, iNumberOfArrayElements);

	  // Comparison

	  const float epsilon = 0.000001f;
	  int breakValue = -1;
	  bool bAccuracy = true;

	  for (int i = 0; i < iNumberOfArrayElements; i++)
	  {
		  float val1 = gold[i];
		  float val2 = hostOutput[i];

		  if (fabs(val1 - val2) > epsilon)
		  {
			  bAccuracy = false;
			  breakValue = i;
			  break;
		  }
	  }

	  char str[128];
	  if (bAccuracy == false)
		  sprintf(str, "Comparison of CPU and GPU Vector Addition is not within accuracy of 0.000001 at array index %d", breakValue);
	  else
		  sprintf(str, "Comparison of CPU and GPU Vector Addition is within accuracy of 0.000001");

	  // output

	  printf("SUM Output: Array1 begins from 0th index %.6f to %dth index %.6f     \n\n", hostInput1[0], iNumberOfArrayElements - 1, hostInput1[iNumberOfArrayElements - 1]);

	  printf("SUM Output: Array2 begins from 0th index %.6f to %dth index %.6f      \n\n", hostInput2[0], iNumberOfArrayElements - 1, hostInput2[iNumberOfArrayElements - 1]);

	  printf("SUM Output: CUDA Kernel Grid dimension = %d,%d,%d and Block Dimension =%d,%d,%d        \n\n", dimGrid.x, dimGrid.y, dimGrid.z, dimBlock.x, dimBlock.y, dimBlock.z);

	  printf("SUM Output: Output Array begins from 0th index %.6f to %dth index %.6f      \n\n", hostOutput[0], iNumberOfArrayElements - 1, hostOutput[iNumberOfArrayElements - 1]);

	  printf("SUM Output: Time Taken for Vector Addition on CPU = %.6f\n\n", timeOnCPU);
	  
	  printf("SUM Output: Time Taken for Vector Addition on GPU = %.6f\n\n", timeOnGPU);
	  printf("%s\n", str);


	  //Cleanup
	  cleanup();

	  return(0);
}

void fillFloatArrayWithRandomNumbers(float* arr, int len)
{
	//code
	const float fscale = 1.0f / (float)RAND_MAX;

	for (int i = 0; i < len; i++)
	{
		arr[i] = fscale * rand();
	}
}

void vecAddCPU(const float* arr1, const float* arr2, float* out, int len)
{
	//code
	StopWatchInterface* timer = NULL;
	sdkCreateTimer(&timer);
	sdkStartTimer(&timer);

	for (int i = 0; i < len; i++)
	{
		out[i] = arr1[i] + arr2[i];
	}

	sdkStopTimer(&timer);
	timeOnCPU = sdkGetTimerValue(&timer);
	sdkDeleteTimer(&timer);
	timer = NULL;
}

void cleanup(void)
{
	//code

	if (gold)
	{
		free(gold);
		gold = NULL;
	}

	if (deviceOutput)
	{
		hipFree(deviceOutput);
		deviceOutput = NULL;
	}

	if (deviceInput2)
	{
		hipFree(deviceInput2);
		deviceInput2 = NULL;
	}

	if (deviceInput1)
	{
		hipFree(deviceInput1);
		deviceInput1 = NULL;
	}

	if (hostOutput)
	{
		free(hostOutput);
		hostOutput = NULL;
	}

	if (hostInput2)
	{
		free(hostInput2);
		hostInput2 = NULL;
	}

	if (hostInput1)
	{
		free(hostInput1);
		hostInput1 = NULL;
	}
}