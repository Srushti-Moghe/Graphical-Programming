#include <stdio.h>
#include <hip/hip_runtime.h>

const int iNumberOfArrayElements = 5;

float* hostInput1 = NULL;
float* hostInput2 = NULL;
float* hostOutput = NULL;

float* deviceInput1 = NULL;
float* deviceInput2 = NULL;
float* deviceOutput = NULL;

// CUDA kernel
__global__ void vecAddGPU(float* in1, float* in2, float* out, int len)
{
// Code
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < len)
	{
		out[i] = in1[i] + in2[i];
	}
}

// Entry-point Function
int main(void)
{
	// Function Declarations

	void cleanup(void);

	// Variable Declarations

	int size = iNumberOfArrayElements * sizeof(float);

	hipError_t result = hipSuccess;

	// code

// host memory allocation

	hostInput1 = (float*)malloc(size);
	if (hostInput1 == NULL)
	{
		printf("SUM: Host Memory allocation is Failed For hostInput1 array");
		cleanup();
		exit(EXIT_FAILURE);
	}

	hostInput2 = (float*)malloc(size);
	if (hostInput2 == NULL)
	{
		printf("SUM: Host Memory allocation is Failed For hostInput2 array");
		cleanup();
		exit(EXIT_FAILURE);
	}
	
	hostOutput = (float*)malloc(size);
	if (hostOutput == NULL)
	{
		printf("SUM: Host Memory allocation is Failed For hostOutput array");
		cleanup();
		exit(EXIT_FAILURE);
	}

	 // filling values into host arrays
	 hostInput1[0] = 501.0;
	 hostInput1[1] = 502.0;
	 hostInput1[2] = 503.0;
	 hostInput1[3] = 504.0;
	 hostInput1[4] = 505.0;
	
	 hostInput2[0] = 101.0;
	 hostInput2[1] = 102.0;
	 hostInput2[2] = 103.0;
	 hostInput2[3] = 104.0;
	 hostInput2[4] = 105.0;

	 // device memory allocation
	  result = hipMalloc((void**)&deviceInput1, size);
	  if (result != hipSuccess)
	  {
		  printf("SUM: Device Memory allocation isfailed	for deviceInput1 array\n");
		  cleanup();
		  exit(EXIT_FAILURE);
	  }

	  result = hipMalloc((void**)&deviceInput2, size);
	  if (result != hipSuccess)
	  {
		  printf("SUM: Device Memory allocation isfailed	for deviceInput12array.\n");
		  cleanup();
		  exit(EXIT_FAILURE);
	  }

	  result = hipMalloc((void**)&deviceOutput, size);
	  if (result != hipSuccess)
	  {
		  printf("SUM: Device Memory allocation isfailed	for deviceOutput array.\n");
		  cleanup();
		  exit(EXIT_FAILURE);
	  }
	  
	  //Copy data from host arrays into device arrays
	  result = hipMemcpy(deviceInput1, hostInput1, size, hipMemcpyHostToDevice);
	  if (result != hipSuccess)
	  {
		  printf("SUM: Host to Device Data copy is Failed for DeviceInput1 array.\n");
		  cleanup();
		  exit(EXIT_FAILURE);
	  }

	  result = hipMemcpy(deviceInput2, hostInput2, size, hipMemcpyHostToDevice);
	  if (result != hipSuccess)
	  {
		  printf("SUM: Host to Device Data copy is Failed for DeviceInput2 array.\n");
		  cleanup();
		  exit(EXIT_FAILURE);
	  }

	  dim3 dimGrid = dim3(iNumberOfArrayElements, 1, 1);
	  dim3 dimBlock = dim3(1, 1, 1);

	  // CUDA kernel for Vector Addition
	  vecAddGPU <<<dimGrid, dimBlock >>> (deviceInput1, deviceInput2,
		  deviceOutput, iNumberOfArrayElements);

	  // copy data from device array into host array
	  result = hipMemcpy(hostOutput, deviceOutput, size,
		  hipMemcpyDeviceToHost);

	  if (result != hipSuccess)

	  {
		  printf("SUM: Device to Host Data Copy is failed for hostOutput array.\n");
		  cleanup();
		  exit(EXIT_FAILURE);

	  }

	  //Vector addition on host
	  for (int i = 0; i < iNumberOfArrayElements; i++)
	  {
		  printf("SUM Output : %f + %f = %f\n", hostInput1[i], hostInput2[i], hostOutput[i]);
	  }

	  //Cleanup
	  cleanup();

	  return(0);
}

void cleanup(void)
{
	//code
	if (deviceOutput)
	{
		hipFree(deviceOutput);
		deviceOutput = NULL;
	}

	if (deviceInput2)
	{
		hipFree(deviceInput2);
		deviceInput2 = NULL;
	}

	if (deviceInput1)
	{
		hipFree(deviceInput1);
		deviceInput1 = NULL;
	}

	if (hostOutput)
	{
		free(hostOutput);
		hostOutput = NULL;
	}

	if (hostInput2)
	{
		free(hostInput2);
		hostInput2 = NULL;
	}

	if (hostInput1)
	{
		free(hostInput1);
		hostInput1 = NULL;
	}
}