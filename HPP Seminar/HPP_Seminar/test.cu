#include <stdio.h>
#include <hip/hip_runtime.h>

int main(void)
{
	hipDeviceProp_t pCuda;
	int count;

	hipGetDeviceCount(&count);

	for (int i = 0; i < count; i++)
	{
		hipGetDeviceProperties(&pCuda, i);
			printf("Name %s\n", pCuda.name);
			printf("computeCapability %d .%d\n", pCuda.major, pCuda.minor);
	}
	return(0);
}
// build
// nvcc


